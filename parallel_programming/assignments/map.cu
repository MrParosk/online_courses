/* 
Implementing map in CUDA. It squares each element in a matrix.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#define BLOCK_WIDTH 2
#define MATRIX_SIZE 4

void print_matrix(unsigned int* array){
    for(unsigned int i = 0; i < MATRIX_SIZE; i++){
        for(unsigned int j = 0; j < MATRIX_SIZE; j++){
            printf("%03d ", array[j + i * MATRIX_SIZE]);
        }
        printf("\n");
    }
}

__device__ unsigned int square(unsigned int x){
    return x*x;
}

__global__ void map(unsigned int* d_out, unsigned int* d_in){
    unsigned int row = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int col = threadIdx.y + blockIdx.y * blockDim.y;

    if (row >= MATRIX_SIZE || col >= MATRIX_SIZE){
        return;
    }

    unsigned int idx = col + row * MATRIX_SIZE;
    d_out[idx] = square(d_in[idx]);
}

int main(){
    const unsigned int NUM_ELEMENTS = MATRIX_SIZE * MATRIX_SIZE;
    const unsigned int BYTES = NUM_ELEMENTS * sizeof(int);
    
    unsigned int h_in[NUM_ELEMENTS];

    for(unsigned int i = 0; i < NUM_ELEMENTS; i++){
        h_in[i] = i;
    }

    unsigned int h_out[NUM_ELEMENTS];

    unsigned int* d_in;
    unsigned int* d_out;

    hipMalloc((void **) &d_in, BYTES);
    hipMalloc((void **) &d_out, BYTES);

    const dim3 block_size(BLOCK_WIDTH, BLOCK_WIDTH, 1);
    const unsigned int row_blocks = (unsigned int) (MATRIX_SIZE / BLOCK_WIDTH + 1);
    const unsigned int col_blocks = (unsigned int) (MATRIX_SIZE / BLOCK_WIDTH + 1);
    const dim3 grid_size(row_blocks, col_blocks, 1);

    hipMemcpy(d_in, h_in, BYTES, hipMemcpyHostToDevice);
    map<<<grid_size, block_size>>>(d_out, d_in);
    hipMemcpy(h_out, d_out, BYTES, hipMemcpyDeviceToHost);

    printf("Matrix before squaring: \n");
    print_matrix(h_in);
    printf("\n");

    printf("Matrix after squaring: \n");
    print_matrix(h_out);

    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
